#include "hip/hip_runtime.h"
/*
  Author : E. Orliac, SCITAS, EPFL
  Date   : 08.11.2017
  Purpose: 
  Remarks: 
*/

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <>
#include <hip/device_functions.h>
#include <math.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>


#define DB6   6
#define DB3   3

__device__ __constant__ double d_g6[DB6];
__device__ __constant__ double d_h6[DB6];
__device__ __constant__ double d_unity[1];


/*
__inline__ __device__ void reverse_vector(double *data, const int size) {

  unsigned   j;
  double   tmp;

  for(unsigned i=0; i<(size/2); i++) {
    j   = size - i - 1;
    tmp = data[j];
    data[j] = data[i];
    data[i] = tmp;
  }
} 
*/

__device__ void filter(double *out,
		       double *in,  const int n,
		       double *num, const int lnum,
		       double *den, const int lden,
		       const bool reverse) {
  int    i, j;
  double acc;

  if (lden == 1 && den[0] == 1.0) {
    if (reverse) {
      for(i=0; i<n; i++) {
	out[n-1-i] = num[0]*in[n-1-i];
	for(j=1; j<lnum; j++) {
	  if (j<=i) {
	    out[n-1-i] += num[j]*in[n-1-i+j];
	  }
	}
      }
    } else {
      /*
      for(i=0; i<n; i++) {
	out[i] = 0.0;
	for(j=0; j<lnum; j++) {
	  if (j<=i) {
	    out[i] += num[j]*in[i-j];
	  }
	}
      }
      */
      for(i=0; i<n; i+=2) {
	out[i]   = num[0]*in[i];
	out[i+1] = num[0]*in[i+1];
	for(j=1; j<lnum; j++) {
	  if (j<=i) {
	    out[i] += num[j]*in[i-j];
	  }
	  if (j<=i+1) {
	    out[i+1] += num[j]*in[i-j+1];
	  }
	}
      }
    }
  } else if (lnum == 1 && num[0] == 1.0) {

    for(i=0; i<n; i++)
      out[i]   = 0.0;

    if (reverse) {
      for(i=0; i<n; i++) {
	acc = in[n-1-i];
	for(j=1; j<lden; j++) {
	  if (j<=i) {
	    acc -= den[j]*out[i-j];
	  }
	}
	out[i] = acc;
      }
    } else {
      for(i=0; i<n; i++) {
	acc = in[i];
	for(j=1; j<lden; j++) {
	  if (j<=i) {
	    acc -= den[j]*out[i-j];
	  }
	}
	out[i] = acc;
      }
    }
  } else {
    printf("Fatal. Unknown case in kernel:filter.\n");	
    assert(false);
  }
}

__device__ void filter_boundary(double *tmp,
				double *out,
				double *data, const int n,
				double *num,  const int lnum,
				const int lden, 
				double *den1, const int lden1,
				double *den2, const int lden2,
				const int condition) {

  unsigned   j  = 0;
  double   dtmp = 0.0;

  if (condition == 1) {
    filter(out, data, n, num, lnum, d_unity, 1, true);
  } else if (condition == 0) {
    filter(out, data, n, num, lnum, d_unity, 1, false);
  } else {
    printf("Fatal. Unknown case in kernel filter_boundary.\n");
    assert(false);
  }
 
  if (lden == 2) {
 
    /* den1 is causal; den2 is non-causal */
    if (lden1+lden2 > 2) {

      // shiftnc = lden2 - 1
      if (lden2 - 1 != 0) {
	printf("FATAL. Non-zero shiftnc! Need to add missing implementation.\n");
	assert(false);
      }

      if (condition == 0) {
	filter(tmp, out, n, d_unity, 1, den1, lden1, false);
	filter(out, tmp, n, d_unity, 1, den2, lden2, true);
	for (unsigned i=0; i<n; i+=2) {
	  out[i]   *= den2[lden2-1];
	  out[i+1] *= den2[lden2-1];
	}
      } else {
	filter(tmp, out, n, d_unity, 1, den1, lden1, true);
	filter(out, tmp, n, d_unity, 1, den2, lden2, true);

	//EO: combine revert and scaling
	for(unsigned i=0; i<(n/2); i++) {
	  j      = n - i - 1;
	  dtmp   = out[j];
	  out[j] = out[i] * den2[lden2-1];
	  out[i] = dtmp   * den2[lden2-1];
	}
      }
    } else {
      printf("fatal: lden1+lden2 <= 2.");
      assert(false);
    }
  } else {
    printf("fatal: lden not equal to 2.");
    assert(false);
  }
}

__global__ void my_kernel(double* __restrict__ tcIn,
			  const int tcLength,
			  const int voxelNb,
			  double *num,
			  const int lnum,
			  const int lden,
			  double *den1, const int lden1,
			  double *den2, const int lden2,
			  const double lambdaTemp,
			  const double maxeig,
			  const int cost_save,
			  const int nit,
			  const double *noiseFinIn,
			  double *tcOut,
			  double *noiseFinOut) {

  double acc, nv, noise_estimate, precision, t_l, lambda_t;

  double lambda   = 0.0;
  double t        = 1.0;
  double maxeig_  = 1.0 / maxeig;
  double tclsqrt_ = sqrt(1.0 / tcLength);

  double *z   = (double*) malloc(tcLength*sizeof(double));
  assert(z != NULL);
  double *z_l = (double*) malloc(tcLength*sizeof(double));
  assert(z_l != NULL);
  double *z3  = (double*) malloc(tcLength*sizeof(double));
  assert(z3 != NULL);
  double *s   = (double*) malloc(tcLength*sizeof(double));
  assert(s != NULL);
  double *tmp = (double*) malloc(tcLength*sizeof(double));
  assert(tmp != NULL);

  unsigned    tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= voxelNb)
    return;

  unsigned tcinid = tid * tcLength;

  /* 
     Compute Daubechies 6 level 1 detailed coefficients
     Replicates Matlab sequence (see MyTemporal.m):
         [coef,  len] = wavedec(s,1,'db3');
         coef(1:len(1)) = [];
   ! Assumes default dwtmode('sym') is set.
   ! Requires even length input signal.
  */

  // Requires even length input signal
  assert(tcLength%2==0);
  
  // Number of detailed coefficients to be computed
  unsigned nc = floor((tcLength-1)*0.5) + DB3;

  double *coeff = (double*) malloc(nc * sizeof(double));
  assert(coeff != NULL);

  // Number of coefficients that do not depend on padded values
  unsigned n1 = nc - 4;

  coeff[0]  = 0.0;
  coeff[0] += d_g6[0] * tcIn[tcinid + 3];
  coeff[0] += d_g6[1] * tcIn[tcinid + 2];
  coeff[0] += d_g6[2] * tcIn[tcinid + 1];
  coeff[0] += d_g6[3] * tcIn[tcinid];
  coeff[0] += d_g6[4] * tcIn[tcinid];
  coeff[0] += d_g6[5] * tcIn[tcinid + 1];

  coeff[1]  = 0.0;
  coeff[1] += d_g6[0] * tcIn[tcinid + 1];
  coeff[1] += d_g6[1] * tcIn[tcinid];
  coeff[1] += d_g6[2] * tcIn[tcinid];
  coeff[1] += d_g6[3] * tcIn[tcinid + 1];
  coeff[1] += d_g6[4] * tcIn[tcinid + 2];
  coeff[1] += d_g6[5] * tcIn[tcinid + 3];

  for (unsigned i=0; i < n1; i++) {
    coeff[i+2] = 0.0;
    for (unsigned j=0; j<DB6; j++) {
      coeff[i+2] += d_g6[j] * tcIn[tcinid + 2*i + j];
    }
  }

  coeff[n1+2]  = 0.0;
  coeff[n1+2] += d_g6[0] * tcIn[tcinid + tcLength - 4];
  coeff[n1+2] += d_g6[1] * tcIn[tcinid + tcLength - 3];
  coeff[n1+2] += d_g6[2] * tcIn[tcinid + tcLength - 2];
  coeff[n1+2] += d_g6[3] * tcIn[tcinid + tcLength - 1];
  coeff[n1+2] += d_g6[4] * tcIn[tcinid + tcLength - 1];
  coeff[n1+2] += d_g6[5] * tcIn[tcinid + tcLength - 2];

  coeff[n1+3]  = 0.0;
  coeff[n1+3] += d_g6[0] * tcIn[tcinid + tcLength - 2];
  coeff[n1+3] += d_g6[1] * tcIn[tcinid + tcLength - 1];
  coeff[n1+3] += d_g6[2] * tcIn[tcinid + tcLength - 1];
  coeff[n1+3] += d_g6[3] * tcIn[tcinid + tcLength - 2];
  coeff[n1+3] += d_g6[4] * tcIn[tcinid + tcLength - 3];
  coeff[n1+3] += d_g6[5] * tcIn[tcinid + tcLength - 4];

  thrust::sort(thrust::seq, coeff, coeff+nc);

  double median = 0.0;
  const size_t lhs = (nc - 1) * 0.5;
  const size_t rhs = nc * 0.5;
  if (lhs == rhs) {
    median = coeff[lhs];
  } else {
    median = (coeff[lhs] + coeff[rhs]) * 0.5;
  }

  for(unsigned i=0; i<nc; i++) {
    coeff[i] = fabs(coeff[i] - median);
  }

  thrust::sort(thrust::seq, coeff, coeff+nc);
  if (lhs == rhs) {
    median = coeff[lhs];
  } else {
    median = (coeff[lhs] + coeff[rhs]) * 0.5;
  }

  free(coeff);

  __syncthreads();


  lambda_t       = median * lambdaTemp;
  noise_estimate = lambda_t;
  precision      = noise_estimate * 1E-5;

  if (noiseFinIn[tid] == 0.0) {
    //printf("case 0.0 in cuda: lambda = %15.10f\n", lambda_t);
    lambda = lambda_t;
  } else {
    //printf("case != 0.0 in cuda: lambda = %15.10f (vox %i)\n", noiseFinIn[tid], tid);
    lambda = noiseFinIn[tid];
  }

  for (unsigned i=0; i<tcLength; i+=2) {
    s[i]   = 0.0;
    s[i+1] = 0.0;
    z[i]   = tcIn[tcinid + i];
    z[i+1] = tcIn[tcinid + i + 1];
  }

  filter_boundary(tmp, &tcOut[tcinid], &tcIn[tcinid], tcLength, num, lnum, lden, den1, lden1, den2, lden2, 0);

  nv = 0.0;

  for (unsigned k=0; k<nit; k++) {
    
    for(unsigned i=0; i<tcLength; i++)
      z_l[i] = z[i];

    filter_boundary(tmp, z,  s, tcLength, num, lnum, lden, den1, lden1, den2, lden2, 1);
    filter_boundary(tmp, z3, z, tcLength, num, lnum, lden, den1, lden1, den2, lden2, 0);

    for (unsigned i=0; i<tcLength; i++) {
      z[i] = maxeig_ * (tcOut[tcinid+i]/lambda - z3[i]) + s[i];
      if (z[i] >  1.0) z[i] =  1.0;
      if (z[i] < -1.0) z[i] = -1.0;
    }
    
    t_l = t;
    t   = 0.5 + sqrt(0.25 + t*t);
    double ct = (t_l - 1.0)/t;

    for (unsigned i=0; i<tcLength; i++)
      s[i] = z[i] + ct * (z[i]-z_l[i]);

    nv = 0.0;
    if (cost_save) {
      printf("Fatal. Missing implementation if cost_save == 1 in Temporal_TA_MEX.\n");
      assert(false);
    } else {
      filter_boundary(tmp, z3, z, tcLength, num, lnum, lden, den1, lden1, den2, lden2, 1);
      acc = 0.0;
      for(unsigned i=0; i<tcLength; i++)
	acc += z3[i]*z3[i];
      nv = lambda * sqrt(acc) * tclsqrt_;
    }
    
    if (fabs(nv-noise_estimate) > precision) {
      lambda *= noise_estimate/nv;
    }
  }

  for(unsigned i=0; i<tcLength; i++)
    tcOut[tcinid + i] = tcIn[tcinid + i] - lambda*z3[i];

  noiseFinOut[tid] = nv;

  free(s);
  free(z3);
  free(z_l);
  free(z);
  free(tmp);
}


// Function to set constant memory for the Daubechies db6 coefficients
void set_constant_memory(const double *g6, const double *h6, const double *one) {
  hipMemcpyToSymbol(HIP_SYMBOL(d_g6),    g6,  DB6  * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(d_h6),    h6,  DB6  * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(d_unity), one,        sizeof(double));
}

