
#include <hip/hip_runtime.h>

#include <hip/device_functions.h>
#include <stdio.h>


__global__ void my_kernel(    double *tcin,     const int  tcLength, const int  voxelNb,
			      double *num,      const int  lnum,
			      const int lden,
			      double *den1,     const int  lden1,
			      double *den2,     const int  lden2,
			      double lambdaTemp,
			      double maxeig,
			      const int cost_save,
			      const int nit,
			      const double *noiseFinIn,
			      double *tcout,
			      double *noiseFinOut);

extern void my_kernel_wrapper(dim3   dimGrid,
			      dim3   dimBlock,
			      double *tcin,     const int  tcLength, const int  voxelNb,
			      double *num,      const int  lnum,
			      const int lden,
			      double *den1,     const int  lden1,
			      double *den2,     const int  lden2,
			      double lambdaTemp,
			      double maxeig,
			      const int cost_save,
			      const int nit,
			      const double *noiseFinIn,
			      double *tcout,
			      double *noiseFinOut) {
  my_kernel<<<dimGrid, dimBlock>>>(tcin, tcLength, voxelNb,
				   num,  lnum,
				   lden,
				   den1, lden1,
				   den2, lden2,
				   lambdaTemp,
				   maxeig,
				   cost_save,
				   nit,
				   noiseFinIn,
				   tcout,
				   noiseFinOut);

  return;
}
